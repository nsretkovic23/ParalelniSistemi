﻿
// ReSharper disable All
#include "hip/hip_runtime.h"


#include <iostream>
#define N 10
#define SCALAR 2


//  Izracunati izraz A - B*x, A i B su vektori, x skalar, napisati i kod za testiranje rezultata

__host__ void printVector(int* vector, const char* vecName)
{
	using std::cout;

	cout << "Vector " << vecName << ": {";
	for(int i = 0; i < N; ++i)
	{
		cout << vector[i];
		if (i != N - 1)
			cout << ", ";
	}
	cout << "}\n";
}

__global__ void calculateOnDevice(int* result, int* a, int* b, int scalar)
{
	int id = threadIdx.x;

	if(id < N)
		result[id] = a[id] - b[id] * scalar;
}

__host__ void calculateOnHost(int* result, int* a, int* b, int scalar)
{
	using std::cout;

	for(int i = 0; i < N; ++i)
	{
		result[i] = a[i] - b[i] * scalar;
	}
}

__host__ int compareCalculationsOnHost(int* host_result, int* device_result)
{
	using std::cout;

	for(int i = 0; i < N; ++i)
	{
		if (host_result[i] != device_result[i])
			return 0;
	}

	return 1;
}


int main()
{
	using  std::cout;

	int* host_a, *host_b, *host_res;
	int* dev_a, *dev_b, *dev_res;

	int* host_calculated_res;

	// HACK: Hardcoded scalar
	int scalar = 5;

	host_a = (int*)malloc(N * sizeof(int));
	host_b = (int*)malloc(N * sizeof(int));
	host_res = (int*)malloc(N * sizeof(int));
	host_calculated_res = (int*)malloc(N * sizeof(int));

	for(int i = 0; i < N; ++i)
	{
		host_a[i] = (i + 1) * 2;
		host_b[i] = (i + 1) * 3;
	}

	printVector(host_a, "A");
	printVector(host_b, "B");

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_res, N * sizeof(int));

	hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, N * sizeof(int), hipMemcpyHostToDevice);

	calculateOnDevice <<<1,N>>> (dev_res, dev_a, dev_b, SCALAR);

	hipMemcpy(host_res, dev_res, N * sizeof(int), hipMemcpyDeviceToHost);

	calculateOnHost(host_calculated_res, host_a, host_b, SCALAR);

	printVector(host_res, "RESULT");
	printVector(host_calculated_res, "RES_SEQUENTIALLY");

	if (compareCalculationsOnHost(host_calculated_res, host_res) > 0)
		cout << "\nCalculations are correct!!!";
	else
		cout << "\nIncorrect calculations";

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_res);
	free(host_a);
	free(host_b);
	free(host_res);

	return 0;
}

